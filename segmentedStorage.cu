#include "queueInterface.cuh"
#include "queueHelpers.cuh"
#include "segmentedStorage.cuh"

void (*SegmentedStorage::pReinitStorage)() = 0;

__device__ void* storage = NULL;

void* SegmentedStorage::StoragePointer = 0;

void SegmentedStorage::destroyStorage()
{
	if(StoragePointer != 0)
		CUDA_CHECKED_CALL(hipFree(&StoragePointer));
	StoragePointer = 0;
	pReinitStorage = 0;
}

void SegmentedStorage::checkReinitStorage()
{
	if(pReinitStorage != 0)
		pReinitStorage();
}

