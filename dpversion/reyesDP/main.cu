#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <string.h>
//#include <windows.h>

// #include "__pikoDefines.h"

#include "reyesPipe.h"
#include "__pikoCompiledPipe.cuh"

#ifdef __PIKOC_HOST__

//#include <GL/glut.h>

#include <piko/builtinTypes.h>
#include "host_math.h"
#include "pikoTypes.h"
#include "FPSMeter.h"

// pikoc does not work well with assimp, so it will not be included when pikoc runs
#ifndef __PIKOC__
#include "sceneParser.h"
#endif  // __PIKOC__


using namespace std;

#define PATCH_BUFFER_SIZE 6000

// ----------------------------------------
// function prototypes
// ----------------------------------------
void init(int argc, char* argv[]);
void initScreen(int W, int H);
void initScene();
void initPipe();
void display();
void destroyApp();
void doPerfTest(int n_runs = 10);
void runPipe();
void pipelineTest();
void resetDepthBuffer();
void printDepthBuffer();

// camera helper functions here
void buildProjectionMatrix();
void buildLookAt();

void glhPerspectivef2(float *matrix, float fovyInDegrees, float aspectRatio,
                      float znear, float zfar);
void glhFrustumf2(float *matrix, float left, float right, float bottom, float top,
                  float znear, float zfar);

void loadPatchBuffer(int start, int end);
// ----------------------------------------
// global variables
// ----------------------------------------


// camera angles
float theta, phi, camdist;

#ifndef __PIKOC__
// main scene
scene sMain;
#endif // __PIKOC__

piko_patch* patchBuffer = NULL;

ReyesPipe piko_pipe;

// state
ConstantState pipelineConstantState;
MutableState pipelineMutableState;

int numPatches;

int main(int argc, char* argv[])
{
	hipSetDevice(1);
  //glutInit(&argc, argv);
  //initScreen(640, 480);
  initScreen(1280, 720);
  //initScreen(1024, 768);
  initScene();
  initPipe();
  display();
  //glutDisplayFunc(display);
  //doPerfTest(5);
  //atexit(destroyApp);
  //glutMainLoop();
}

cvec4f matmultfloat4(float * mvpMat, cvec4f v)
{
  cvec4f outRes;
  (outRes).x = mvpMat[0] * v.x + mvpMat[4] * v.y + mvpMat[8 ] * v.z + mvpMat[12] * v.w;
  (outRes).y = mvpMat[1] * v.x + mvpMat[5] * v.y + mvpMat[9 ] * v.z + mvpMat[13] * v.w;
  (outRes).z = mvpMat[1] * v.x + mvpMat[6] * v.y + mvpMat[10] * v.z + mvpMat[14] * v.w;
  (outRes).w = mvpMat[3] * v.x + mvpMat[7] * v.y + mvpMat[11] * v.z + mvpMat[15] * v.w;
  return outRes;
}


void initScreen(int W, int H){
  #ifndef __PIKOC__
  sMain.cam().W() = W;
  sMain.cam().H() = H;
#endif // __PIKOC__

  pipelineConstantState.screenSizeX = W;
  pipelineConstantState.screenSizeY = H;

  //glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
  //glutInitWindowSize(W,H);
  //glutCreateWindow("Reyes Pipeline");
  //glClearColor(0.0f, 0.0f, 0.2f, 1.0f);
}


void display()
{
  // update state

  buildProjectionMatrix();
  resetDepthBuffer();

  printf("running display\n");

  piko_pipe.prepare();
  
  printf("After prepare\n");
  
  piko_pipe.run_single();
  
  printf("After single\n");

  unsigned* data =  piko_pipe.pikoScreen.getData();
  
  int x = pipelineConstantState.screenSizeX, y = pipelineConstantState.screenSizeY;
  FILE *fp = fopen("result", "w");
  for(int i=0; i<y; i++)
  {
	  for(int j=0; j<x*4; j++)
	  {
		  fprintf(fp, "%d", ((char*)data)[i*x*4 + j]);
		  if(j != x*4 - 1)
		  {
			  fprintf(fp, "\t");
		  }
	  }
	  if(i != y - 1)
	  {
		fprintf(fp, "\n");
	  }
  }
  fclose(fp);
  
  return;

  //glDrawPixels(pipelineConstantState.screenSizeX, pipelineConstantState.screenSizeY, GL_RGBA, GL_UNSIGNED_BYTE, data);
  //glutSwapBuffers();

  // for(int i=0; i< pipelineConstantState.screenSizeX * pipelineConstantState.screenSizeY; i++)
  // {
  //   if(data[i] != 0)
  //     printf("%d: %x\n", i, data[i]);
  // }
}

void doPerfTest(int n_runs)
{
  printf("Running perf test...\n");

  buildProjectionMatrix();
  resetDepthBuffer();
  piko_pipe.prepare();
  piko_pipe.run_single();

  Stopwatch mywatch;

  mywatch.Reset();
  for(int run = 0; run < n_runs; run++)
  {
	  printf("Start %d ----------  ", run);
    buildProjectionMatrix();
    resetDepthBuffer();
    piko_pipe.prepare();
	printf("end\n");
  }
  float prepTime = mywatch.GetTime();

  mywatch.Reset();
  for(int run = 0; run < n_runs; run++)
  {
	  printf("Start %d ----------  ", run);
	  fflush(NULL);
    buildProjectionMatrix();
    resetDepthBuffer();
    piko_pipe.prepare();
    piko_pipe.run_single();
	printf("end\n");
  }
  float fullrunTime = mywatch.GetTime();

  float total_time_to_ms = 1000.0f / (float) n_runs;

  printf("Prep time     = %0.2f ms\n", total_time_to_ms * (prepTime));
  printf("Full run time = %0.2f ms\n", total_time_to_ms * (fullrunTime));
  printf("Raster time   = %0.2f ms\n", total_time_to_ms * (fullrunTime - prepTime));
}

void initScene(){
  // the scene will only be compiled when going through gcc and not pikoc
  // parse scene file
  sceneParser scp;

  int nT, nV, nP;

  buildProjectionMatrix();


  scp.parseFile("./", "bezteapot.scene", &sMain);

  printf("Flattening scene assets: "); fflush(stdout);
  sMain.flatten(nT,nV, nP);
  printf("T: %d, V: %d P:%d\n", nT, nV, nP);
  numPatches = nP;

  // create the final matrix
  // FIXME: perhaps this is flipped?
  //HOST::matmult4x4(pipelineConstantState.projMatrix, pipelineConstantState.viewMatrix,pipelineConstantState.viewProjMatrix );
  //HOST::matmult4x4(pipelineConstantState.viewMatrix, pipelineConstantState.projMatrix, pipelineConstantState.viewProjMatrix);

  sMain.cam().display();

}

void initPipe()
{
  // build the state from the scene
  // camera& cam = sMain.cam();
  // pipelineConstantState.camera_eye = cam.eye();
  // pipelineConstantState.camera_target = cam.target();
  // pipelineConstantState.camera_up = cam.up();
  // if(sMain.lights().size() > 0) {
  //   pipelineConstantState.lightPos = sMain.lights()[0].pos();
  //   pipelineConstantState.lightColor = sMain.lights()[0].dif();
  // }
  // else {
  //   // some default light that might suck
  //   pipelineConstantState.lightPos = gencvec3f(1.0,1.0,1.0);
  //   pipelineConstantState.lightColor = gencvec3f(1.0,1.0,1.0);
  // }

  int numLoadPatches = numPatches;
  loadPatchBuffer(0,numLoadPatches);
  resetDepthBuffer();
  piko_pipe.allocate(pipelineConstantState, pipelineMutableState, patchBuffer, numLoadPatches);
}

// void runPipe()
// {
//   int count = 1;
//   ReyesPipe p;
//   p.run(pipelineConstantState,patchBuffer, 1);
// }

void getPerspectiveMat(float *mat, float fovy, 
		float aspect, float n, float f)
{
	float l, r, b, t;
	float pi = 4.0 * atan(1);
	float jiao = fovy * pi / 360;
	t = n * tanf(jiao);
	b = -t;
	l = b * aspect;
	r = t * aspect;
	
	memset(mat, 0, sizeof(float)*16);
	mat[0] = (2.0f * n) / (r-l);
	mat[2] = (r+l)/(r-l);
	mat[5] = (2.0f * n) / (t-b);
	mat[6] = (t+b)/(t-b);
	mat[10] = -(f+n)/(f-n);
	mat[11] = -(2.0f*f*n)/(f-n);
	mat[14] = -1.0;
}

void getLookUpMat(float *res, float ex, float ey, float ez,
	float tx, float ty, float tz, float ux, float uy, float uz)
{
	float fx = tx - ex;
	float fy = ty - ey;
	float fz = tz - ez;
	float flength = sqrt(fx*fx + fy*fy + fz*fz);
	fx /= flength;
	fy /= flength;
	fz /= flength;
	
	float sx, sy, sz;
	sx = fy*uz - fz*uy;
	sy = fz*ux - fx*uz;
	sz = fx*uy - fy*ux;
	float slength = sqrt(sx*sx + sy*sy + sz*sz);
	sx /= slength;
	sy /= slength;
	sz /= slength;
	
	ux = sy*fz - sz*fy;
	uy = sz*fx - sx*fz;
	uz = sx*fy - sy*fx;
	
	float mat[16];
	memset(mat, 0, sizeof(float)*16);
	mat[0] = 1;
	mat[5] = 1;
	mat[10] = 1;
	mat[15] = 1;
	
	mat[0] = sx;
	mat[1] = sy;
	mat[2] = sz;
	mat[4] = ux;
	mat[5] = uy;
	mat[6] = uz;
	mat[8] = -fx;
	mat[9] = -fy;
	mat[10] = -fz;
	
	float trans[16];
	memset(trans, 0, sizeof(float)*16);
	trans[0] = 1;
	trans[5] = 1;
	trans[10] = 1;
	trans[15] = 1;
	trans[3] = -ex;
	trans[7] = -ey;
	trans[11] = -ez;
	
	memset(res, 0, sizeof(float)*16);
	for(int i=0; i<4; i++)
	{
		for(int j=0; j<4; j++)
		{
			for(int k=0; k<4; k++)
			{
				res[i*4 + j] += mat[i*4 + k] * trans[k*4 + j];
			}
		}
	}
	
	
}

void buildProjectionMatrix()
{
	
  camera& cam = sMain.cam();
	float viewmat[16], mat[16], resProj[16], resView[16];
  
	//glMatrixMode(GL_PROJECTION);

  //glLoadIdentity();
	memset(viewmat, 0, sizeof(float)*16);
  viewmat[0] = 1;
  viewmat[5] = 1;
  viewmat[10] = 1;
  viewmat[15] = 1;
	getPerspectiveMat(mat, cam.fovyDeg(), cam.aspect(), cam.zNear(), cam.zFar());
  //gluPerspective(cam.fovyDeg(), cam.aspect(), cam.zNear(), cam.zFar());
  //glGetFloatv(GL_PROJECTION_MATRIX, viewmat);
	memset(resProj, 0, sizeof(resProj));
	for(int i=0; i<4; i++)
	{
		for(int j=0; j<4; j++)
		{
			for(int k=0; k<4; k++)
			{
				// res和viewmat是列优先，mat是行优先
				resProj[j*4+i] += viewmat[k*4+i]*mat[k*4+j];
			}
		}
	}
	//glLoadMatrixf(resProj);
  
  
  //glMatrixMode(GL_MODELVIEW);
  //glLoadIdentity();
  
  memset(viewmat, 0, sizeof(float)*16);
  viewmat[0] = 1;
  viewmat[5] = 1;
  viewmat[10] = 1;
  viewmat[15] = 1;
  
  float lu[16];
	getLookUpMat(lu, cam.eye().x,    cam.eye().y,    cam.eye().z,
      cam.target().x, cam.target().y, cam.target().z,
      cam.up().x,     cam.up().y,     cam.up().z);
	memset(resView, 0, sizeof(resView));
	for(int i=0; i<4; i++)
	{
		for(int j=0; j<4; j++)
		{
			for(int k=0; k<4; k++)
			{
				// res和viewmat是列优先，lu是行优先
				resView[j*4+i] += viewmat[k*4+i]*lu[k*4+j];
			}
		}
	}
	//glLoadMatrixf(resView);
	
	
  /*
  gluLookAt(
      cam.eye().x,    cam.eye().y,    cam.eye().z,
      cam.target().x, cam.target().y, cam.target().z,
      cam.up().x,     cam.up().y,     cam.up().z);
	  */
	  
  /*
  glGetFloatv(GL_MODELVIEW_MATRIX, viewmat);
	for(int i=0; i<4; i++)
	{
		for(int j=0; j<4; j++)
		{
			printf("%.2f\t", viewmat[i*4+j]);
		}
		printf("\n");
	}
	printf("\n");
	printf("\n");
	
	
	for(int i=0; i<4; i++)
	{
		for(int j=0; j<4; j++)
		{
			printf("%.2f\t", lu[i*4+j]);
		}
		printf("\n");
	}
	printf("\n");
	printf("\n");
	*/

  //glMatrixMode(GL_PROJECTION);
  //glPushMatrix();
  // glGetFloatv(GL_MODELVIEW_MATRIX, pipelineConstantState.viewMatrix);
  memcpy(pipelineConstantState.viewMatrix, resView, sizeof(float)*16);
  //glMultMatrixf(pipelineConstantState.viewMatrix);
  float newProj[16];
  memset(newProj, 0, sizeof(newProj));
  for(int i=0; i<4; i++)
  {
	  for(int j=0; j<4; j++)
	  {
		  for(int k=0; k<4; k++)
		  {
			  newProj[j*4 + i] += resProj[k*4+i] * resView[j*4 + k];
		  }
	  }
  }
    memcpy(pipelineConstantState.viewProjMatrix, newProj, sizeof(float)*16);

  //glGetFloatv(GL_PROJECTION_MATRIX, pipelineConstantState.viewProjMatrix);
  //glPopMatrix();  
  
  
  // printf("final projection matrix:\n");
  // for(int i=0; i<16; i++) {
  //   if (i%4 ==0) printf("\n");
  //     printf("%f ", pipelineConstantState.viewProjMatrix[i]);
  // } printf("\n");

  //   printf("final modelview matrix:\n");
  // for(int i=0; i<16; i++) {
  //   if (i%4 ==0) printf("\n");
  //     printf("%f ", pipelineConstantState.viewMatrix[i]);
  // } printf("\n");
}



void loadPatchBuffer(int start, int end) {
  // lazy create
  if(patchBuffer == NULL) {
    patchBuffer = new piko_patch[PATCH_BUFFER_SIZE];
  }

  int size = end - start;

  if (size <=0) return;

  int counter = 0;
  printf("\nfetching patches from %d to %d\n", start, end);
  for(int i=start; i<end; i++) {
    for(int j=0; j<16; j++)
    {
      patchBuffer[counter].CP[j] = sMain._flatPatches[i*16+j];
      //printf("flat patch: ");
      //disp4(patchBuffer[counter].CP[j]);
      //disp4(sMain._flatPatches[i*16+j]);
      //printf("\n");
    }
    patchBuffer[counter].numSplits = 0;     // all patches begin with zero splits
    //patchBuffer[counter].id = counter;
    //patchBuffer[counter].bbmin.x = 99999.0f;
    //patchBuffer[counter].bbmin.y = 99999.0f;

    //patchBuffer[counter].bbmax.x = -99999.0f;
    //patchBuffer[counter].bbmax.y = -99999.0f;
    counter++;
  }
}

void pipelineTest()
{
  // test out parts of the pipeline here
  cvec4f point = patchBuffer[0].CP[0];
  printf("\n\n point:\n");
  disp4(point);
  cvec4f clipPoint = matmultfloat4(pipelineConstantState.viewProjMatrix, point);

  if(clipPoint.w == 0.0f) clipPoint.w = 1.0f;

            clipPoint.x /= clipPoint.w;
            clipPoint.y /= clipPoint.w;
            clipPoint.z /= clipPoint.w;

            clipPoint.x = (clipPoint.x+1.0) * 0.5 * pipelineConstantState.screenSizeX;
            clipPoint.y = (clipPoint.y+1.0) * 0.5 * pipelineConstantState.screenSizeY;
    printf("\n");
    disp4(clipPoint);
    printf("\n");
}

void resetDepthBuffer() {
  int nPixels = pipelineConstantState.screenSizeX * pipelineConstantState.screenSizeY;
  for(int i = 0; i < nPixels; i++) {
    pipelineMutableState.zBuffer[i] = 1.0f;
  }
}

void printDepthBuffer() {
  int nPixels = pipelineConstantState.screenSizeX * pipelineConstantState.screenSizeY;
  for(int i = 0; i < nPixels; i++) {
    printf("%f\n", pipelineMutableState.zBuffer[i]);
  }
}

void destroyApp()
{
  piko_pipe.destroy();
}

#endif // __PIKOC_HOST__
