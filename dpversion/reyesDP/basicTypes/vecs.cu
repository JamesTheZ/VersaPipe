#include "hip/hip_runtime.h"
#include "vecs.h"

// 4D
vec4f operator+(const vec4f& v1, const vec4f &v2)
{
	return vec4f(v1.peekx()+v2.peekx(),
                 v1.peeky()+v2.peeky(),
                 v1.peekz()+v2.peekz(),
				 v1.peekw()+v2.peekw());
}

vec4f operator-(const vec4f& v1, const vec4f &v2)
{
	return vec4f(v1.peekx()-v2.peekx(),
                 v1.peeky()-v2.peeky(),
                 v1.peekz()-v2.peekz(),
				 v1.peekw()-v2.peekw());
}
vec4f operator*(const vec4f& v1, const vec4f &v2)
{
	return vec4f(v1.peekx()*v2.peekx(),
                 v1.peeky()*v2.peeky(),
                 v1.peekz()*v2.peekz(),
				 v1.peekw()*v2.peekw());
}


vec4f operator*(const float& k, const vec4f& v1){
    return vec4f(v1.peekx()*k,
                 v1.peeky()*k,
                 v1.peekz()*k,
				 v1.peekw()*k);
}

vec4f operator*(const vec4f& v1, const float& k){
    return vec4f(v1.peekx()*k,
                 v1.peeky()*k,
                 v1.peekz()*k,
				 v1.peekw()*k);
}



// 3D

vec3f operator+(const vec3f& v1, const vec3f &v2){
    return vec3f(v1.peekx()+v2.peekx(),
                 v1.peeky()+v2.peeky(),
                 v1.peekz()+v2.peekz());
}

vec3f operator-(const vec3f& v1, const vec3f &v2){
    return vec3f(v1.peekx()-v2.peekx(),
                 v1.peeky()-v2.peeky(),
                 v1.peekz()-v2.peekz());
}

vec3f operator*(const vec3f& v1, const float k){
    return vec3f(v1.peekx()*k,
                 v1.peeky()*k,
                 v1.peekz()*k);
}

vec3f operator*(const float k, const vec3f& v1){
    return vec3f(v1.peekx()*k,
                 v1.peeky()*k,
                 v1.peekz()*k);
}

vec3f operator*(const vec3f& v1, const vec3f &v2)
{
	return vec3f(v1.peekx()*v2.peekx(),
                 v1.peeky()*v2.peeky(),
                 v1.peekz()*v2.peekz());
}

float dot(const vec3f& v1, const vec3f& v2){
    return ((v1.peekx()*v2.peekx())+
            (v1.peeky()*v2.peeky())+
            (v1.peekz()*v2.peekz()));
}

float dot(const vec4f& v1, const vec3f& v2){
    return ((v1.peekx()*v2.peekx())+
            (v1.peeky()*v2.peeky())+
            (v1.peekz()*v2.peekz()));
}

float dot(const vec3f& v1, const vec4f& v2){
    return ((v1.peekx()*v2.peekx())+
            (v1.peeky()*v2.peeky())+
            (v1.peekz()*v2.peekz()));
}

float dot(const vec4f& v1, const vec4f& v2){
    return ((v1.peekx()*v2.peekx())+
            (v1.peeky()*v2.peeky())+
            (v1.peekz()*v2.peekz()));
}


// 2D


vec2f operator+(const vec2f& v1, const vec2f &v2){
    return vec2f(v1.peekx()+v2.peekx(),
                 v1.peeky()+v2.peeky());
}

vec2f operator-(const vec2f& v1, const vec2f &v2){
    return vec2f(v1.peekx()-v2.peekx(),
                 v1.peeky()-v2.peeky());
}

vec2f operator*(const vec2f& v1, const float k){
    return vec2f(v1.peekx()*k,
                 v1.peeky()*k);
}

vec2f operator*(const float k, const vec2f& v1){
    return vec2f(v1.peekx()*k,
                 v1.peeky()*k);
}

float dot(const vec2f& v1, const vec2f& v2){
    return ((v1.peekx()*v2.peekx())+
            (v1.peeky()*v2.peeky()));
}





vec2i operator+(const vec2i& v1, const vec2i &v2){
    return vec2i(v1.peekx()+v2.peekx(),
                 v1.peeky()+v2.peeky());
}

vec2i operator-(const vec2i& v1, const vec2i &v2){
    return vec2i(v1.peekx()-v2.peekx(),
                 v1.peeky()-v2.peeky());
}

vec2i operator*(const vec2i& v1, const int k){
    return vec2i(v1.peekx()*k,
                 v1.peeky()*k);
}

vec2i operator*(const int k, const vec2i& v1){
    return vec2i(v1.peekx()*k,
                 v1.peeky()*k);
}

int dot(const vec2i& v1, const vec2i& v2){
    return ((v1.peekx()*v2.peekx())+
            (v1.peeky()*v2.peeky()));
}


// misc math
bool baryCentricTriangle(vec2f p, vec4f v1, vec4f v2, vec4f v3, float &u, float &v, float &r)
{
	float x1mx3 = v1.x() - v3.x();
	float x2mx3 = v2.x() - v3.x();
	float y1my3 = v1.y() - v3.y();
	float y2my3 = v2.y() - v3.y();

	float det = (x1mx3 * y2my3) - (y1my3*x2mx3);

	float pxmx3 = p.x() - v3.x();
	float pymy3 = p.y() - v3.y();

	if(det == 0.0 || det == -0.0) return false;

	u = (y2my3*pxmx3 + x2mx3*-1*pymy3)/det;
	v = (y1my3*-1*pxmx3 + x1mx3*pymy3)/det;
	r = 1-u-v;

	if (u > 1.0f || v > 1.0f || r > 1.0f) return false;
	if (u < 0.0f || v < 0.0f || r < 0.0f) return false;

	return true;
}

bool baryCentricTriangle(vec2f p, vec3f v1, vec3f v2, vec3f v3, float &u, float &v, float &r)
{
	float x1mx3 = v1.x() - v3.x();
	float x2mx3 = v2.x() - v3.x();
	float y1my3 = v1.y() - v3.y();
	float y2my3 = v2.y() - v3.y();

	float det = (x1mx3 * y2my3) - (y1my3*x2mx3);

	float pxmx3 = p.x() - v3.x();
	float pymy3 = p.y() - v3.y();

	if(det == 0.0 || det == -0.0) return false;

	u = (y2my3*pxmx3 + x2mx3*-1*pymy3)/det;
	v = (y1my3*-1*pxmx3 + x1mx3*pymy3)/det;
	r = 1-u-v;

	if (u > 1.0f || v > 1.0f || r > 1.0f) return false;
	if (u < 0.0f || v < 0.0f || r < 0.0f) return false;

	return true;
}

void clamp(vec3f & v, float min, float max)
{
	if(v.peekx() > max) v.x() = max;
	if(v.peekx() < min) v.x() = min;

	if(v.peeky() > max) v.y() = max;
	if(v.peeky() < min) v.y() = min;

	if(v.peekz() > max) v.z() = max;
	if(v.peekz() < min) v.z() = min;
}

void matmultvec4f(const float * matrix, const vec4f& v, vec4f& ssv)
{
	ssv.x() = matrix[0] * v.peekx() + matrix[4] * v.peeky() +  matrix[8] * v.peekz() + matrix[12] * v.peekw();
	ssv.y() = matrix[1] * v.peekx() + matrix[5] * v.peeky() +  matrix[9] * v.peekz() + matrix[13] * v.peekw();
	ssv.z() = matrix[2] * v.peekx() + matrix[6] * v.peeky() + matrix[10] * v.peekz() + matrix[14] * v.peekw();
	ssv.w() = matrix[3] * v.peekx() + matrix[7] * v.peeky() + matrix[11] * v.peekz() + matrix[15] * v.peekw();
}
