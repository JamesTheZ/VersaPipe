
#include <hip/hip_runtime.h>
//#include "techniqueMegakernel.cuh"

#ifndef PROC_MAX_NUM
#define PROC_MAX_NUM 64
#endif

#ifndef SM_MAX_NUM
#define SM_MAX_NUM 50
#endif

#ifndef MEGAKERNEL_MAX_PROC_NUM
#define MEGAKERNEL_MAX_PROC_NUM 10
#endif

__device__ void* queuePointers[PROC_MAX_NUM];

namespace Megakernel
{
	__device__ volatile int doneCounter[PROC_MAX_NUM];
	__device__ volatile int endCounter[PROC_MAX_NUM];

	__device__ int maxConcurrentBlocks[PROC_MAX_NUM];
	__device__ volatile int maxConcurrentBlockEvalDone[PROC_MAX_NUM];

	__device__ volatile int sm_flag[PROC_MAX_NUM * SM_MAX_NUM];
	//__device__ int proc_exe_count[PROC_MAX_NUM];
	
	__device__ int block_count[PROC_MAX_NUM * SM_MAX_NUM];

	__device__ int group_done_flag[PROC_MAX_NUM];

	//int numGroups;
	//int procGroupArray[PROC_MAX_NUM];
	__device__ int *procIdArray_global;
	int taskCountArray[PROC_MAX_NUM];

	__device__ int resultCounter[PROC_MAX_NUM];

}

