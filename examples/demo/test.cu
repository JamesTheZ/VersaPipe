#include "hip/hip_runtime.h"
//  Project Whippletree
//  http://www.icg.tugraz.at/project/parallel
//
//  Copyright (C) 2014 Institute for Computer Graphics and Vision,
//                     Graz University of Technology
//
//  Author(s):  Markus Steinberger - steinberger ( at ) icg.tugraz.at
//              Michael Kenzel - kenzel ( at ) icg.tugraz.at
//              Pedro Boechat - boechat ( at ) icg.tugraz.at
//              Bernhard Kerbl - kerbl ( at ) icg.tugraz.at
//              Mark Dokter - dokter ( at ) icg.tugraz.at
//              Dieter Schmalstieg - schmalstieg ( at ) icg.tugraz.at
//
//  Permission is hereby granted, free of charge, to any person obtaining a copy
//  of this software and associated documentation files (the "Software"), to deal
//  in the Software without restriction, including without limitation the rights
//  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
//  copies of the Software, and to permit persons to whom the Software is
//  furnished to do so, subject to the following conditions:
//
//  The above copyright notice and this permission notice shall be included in
//  all copies or substantial portions of the Software.
//
//  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
//  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
//  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
//  THE SOFTWARE.
//

//  Project Whippletree
//  http://www.icg.tugraz.at/project/parallel
//
//  Copyright (C) 2014 Institute for Computer Graphics and Vision,
//                     Graz University of Technology
//
//  Author(s):  Markus Steinberger - steinberger ( at ) icg.tugraz.at
//              Michael Kenzel - kenzel ( at ) icg.tugraz.at
//              Pedro Boechat - boechat ( at ) icg.tugraz.at
//              Bernhard Kerbl - kerbl ( at ) icg.tugraz.at
//              Mark Dokter - dokter ( at ) icg.tugraz.at
//              Dieter Schmalstieg - schmalstieg ( at ) icg.tugraz.at
//
//  Permission is hereby granted, free of charge, to any person obtaining a copy
//  of this software and associated documentation files (the "Software"), to deal
//  in the Software without restriction, including without limitation the rights
//  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
//  copies of the Software, and to permit persons to whom the Software is
//  furnished to do so, subject to the following conditions:
//
//  The above copyright notice and this permission notice shall be included in
//  all copies or substantial portions of the Software.
//
//  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
//  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
//  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
//  THE SOFTWARE.
//

#include <hip/hip_runtime_api.h>

#include <unistd.h>
#include <cxxabi.h>

#include "queueDistLocks.cuh"
#include "queueShared.cuh"
#include "queuingPerProc.cuh"
#include "techniqueMegakernel.cuh"
#include "techniqueKernels.cuh"
#include "techniqueDynamicParallelism.cuh"
#include "segmentedStorage.cuh"

#include "proc.cuh"

#define TIMEVAL_MAX_COUNT 1000000

int *resCount;
__device__ int * resCnt_device[1];
__device__ int localDoneCounter;

struct timeval startTime;
int val[TIMEVAL_MAX_COUNT];
struct timeval tvs[TIMEVAL_MAX_COUNT];

struct timeval timeDelta(struct timeval start, struct timeval end)
{
	struct timeval delta;
	delta.tv_sec = end.tv_sec - start.tv_sec;
	delta.tv_usec = end.tv_usec - start.tv_usec;
	if(delta.tv_usec < 0)
	{
		delta.tv_usec += 1e6;
		delta.tv_sec--;
	}

	return delta;
}

void timePrint(struct timeval time)
{
	printf("%ld.%06ld\n", time.tv_sec, time.tv_usec);
}

void getOutput()
{
	int res = 0;
	int i = 0;
	while(1)
	{
		int curRes = *resCount;
//		printf("curRes: %d, res: %d\n", curRes, res);
		if(curRes != res)
		{
			res = curRes;

			struct timeval curTime;
			gettimeofday(&curTime, NULL);

			val[i] = curRes;
			tvs[i] = timeDelta(startTime, curTime);
	//		if(tvs[i].tv_sec >= 10)
			{
//				printf("%d\t", curRes);
				printf("%f\n", 1.0 * curRes / (tvs[i].tv_sec + tvs[i].tv_usec/1000000.0));

	//			exit(0);
//				timePrint(tvs[i]);
			}

			i++;
		}
		usleep(1);
	}
}

void runTest(int cuda_device)
{
	hipSetDevice(cuda_device);

	// ************************************************
	// Do this before everything
	int h_sm_flags[PROC_MAX_NUM * SM_MAX_NUM];
	memset(h_sm_flags, 0, sizeof(int)*PROC_MAX_NUM*SM_MAX_NUM);

	h_sm_flags[0 * SM_MAX_NUM + 0] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 1] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 2] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 3] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 4] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 5] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 6] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 7] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 8] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 9] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 10] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 11] = 1;
	h_sm_flags[0 * SM_MAX_NUM + 12] = 1;

	h_sm_flags[1 * SM_MAX_NUM + 0] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 1] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 2] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 3] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 4] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 5] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 6] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 7] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 8] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 9] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 10] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 11] = 1;
	h_sm_flags[1 * SM_MAX_NUM + 12] = 1;

	h_sm_flags[2 * SM_MAX_NUM + 0] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 1] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 2] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 3] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 4] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 5] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 6] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 7] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 8] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 9] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 10] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 11] = 1;
	h_sm_flags[2 * SM_MAX_NUM + 12] = 1;

	hipMemcpyToSymbol(HIP_SYMBOL(Megakernel::sm_flag), h_sm_flags, sizeof(int)*PROC_MAX_NUM*SM_MAX_NUM);

	// blocks for each group on each SM
	// [groupId, smid]
	int h_block_count[PROC_MAX_NUM * SM_MAX_NUM];
	for(int i=0; i<PROC_MAX_NUM*SM_MAX_NUM; i++)
	{
		h_block_count[i] = BLOCK_PER_SM_MAX_NUM;
	}

#include "block_count.cuh"

	hipMemcpyToSymbol(HIP_SYMBOL(Megakernel::block_count), h_block_count, sizeof(int)*PROC_MAX_NUM*SM_MAX_NUM);


	/*
	Megakernel::numGroups = 3;
	Megakernel::procGroupArray[0] = 1;
	Megakernel::procGroupArray[1] = 1;
	Megakernel::procGroupArray[2] = 1;
	*/

#ifdef RECURSIVE
	Megakernel::taskCountArray[0] = (int)(COUNT * 4.6 + 0.1);
#else
	Megakernel::taskCountArray[0] = COUNT;
#endif
	Megakernel::taskCountArray[1] = COUNT * BEI_SHU;
	Megakernel::taskCountArray[2] = COUNT * BEI_SHU;

	//Megakernel::initAll();

	hipHostAlloc((void**)&resCount, sizeof(int), hipHostMallocMapped);
	memset(resCount, 0, sizeof(int));
	int *tmpPnt;
	hipHostGetDevicePointer((void **)&tmpPnt, (void*)resCount, 0);
	hipMemcpyToSymbol(HIP_SYMBOL(resCnt_device), &tmpPnt, sizeof(int*));
	int tmpDoneCnt = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(localDoneCounter), &tmpDoneCnt, sizeof(int));

	// ************************************************

	//create everything
#ifdef WHIPPLETREE
	MyTechnique technique;
	technique.init();
#endif


#ifdef HYBRIDPIPE
	Technique_0 technique_0;
	Technique_1 technique_1;
	Technique_2 technique_2;

	technique_0.init();
	technique_1.init();
	technique_2.init();
#endif

	CUDA_CHECKED_CALL(hipDeviceSynchronize());

	printf("After init\n");


	int4 *d, d_h;// = make_int4(1, 0, 1, 2);
	CUDA_CHECKED_CALL(hipMalloc((void**)&d, sizeof(int4)));
	d_h = {1, 2, 3, 4};	
	CUDA_CHECKED_CALL(hipMemcpy(d, &d_h, sizeof(int4), hipMemcpyHostToDevice));


	//hipMalloc((void**)&rstCnt, sizeof(int));
	//hipMalloc((void**)&rstData, sizeof(float)*COUNT*BEI_SHU);

	/*
	   hipMemset(&rstCnt, 0, sizeof(int));
	   hipMemset(rstData, 0, sizeof(float)*COUNT*BEI_SHU);
	   */

	//printf("start insert\n");

	float tmp = 1;
	for(int i=1; i<=COUNT; i++)
	{
		if(tmp > 50)
		{
			tmp = 1;
		}
		float *data_d;
		hipMalloc((void**)&data_d, sizeof(float));
		hipMemcpy(data_d, &tmp, sizeof(float), hipMemcpyHostToDevice);
#ifdef WHIPPLETREE
		technique.insertIntoQueue<InitProc, InitProc::ExpectedData>(1, data_d);
#endif
#ifdef HYBRIDPIPE
		technique_0.insertIntoQueue<InitProc, InitProc::ExpectedData>(1, data_d);
#endif

		tmp += 10;
	}


	hipStream_t stream;
	CUDA_CHECKED_CALL(hipStreamCreate(&stream));
	hipEvent_t a, b;
	CUDA_CHECKED_CALL(hipEventCreate(&a));
	CUDA_CHECKED_CALL(hipEventCreate(&b));
	CUDA_CHECKED_CALL(hipEventRecord(a, stream));

	gettimeofday(&startTime, NULL);
	
	struct timeval beginTime, endTime;
	
	hipDeviceSynchronize();
	gettimeofday(&beginTime, NULL);

	{
#ifdef WHIPPLETREE
		technique.execute(0);
#endif
#ifdef HYBRIDPIPE
		technique_0.executeStreams(0);
		technique_1.executeStreams(0);
		technique_2.executeStreams(0);
#endif
	}

	//getOutput();

	CUDA_CHECKED_CALL(hipDeviceSynchronize());

	gettimeofday(&endTime, NULL);
	struct timeval delta = timeDelta(beginTime, endTime);
	timePrint(delta);

	{
//		Megakernel::InitProcs<MyTechnique, 0, 3>::execute(technique, 0);
//		Megakernel::ExecuteProcs<MyTechnique, 0, 3>::execute(technique, 0);
	}

	{
		//Megakernel::InitProcs<OtherTechnique, 0, 1>::execute(tec_2, 0);
	}


	/*
	CUDA_CHECKED_CALL(hipEventRecord(b, stream));
	CUDA_CHECKED_CALL(hipEventSynchronize(b));
	float time;
	CUDA_CHECKED_CALL(hipEventElapsedTime(&time, a, b));
	time /= 1000.0;
	CUDA_CHECKED_CALL(hipEventDestroy(a));
	CUDA_CHECKED_CALL(hipEventDestroy(b));
	CUDA_CHECKED_CALL(hipStreamDestroy(stream));
	*/

	//printf("run completed in %fs\n", time);
	

	/*
	   int realCnt = 0;
	   hipMemcpy(&realCnt, &rstCnt, sizeof(int), hipMemcpyDeviceToHost);
	   printf("The count is : %d\n", realCnt);
	   float rstHstData[COUNT*BEI_SHU];
	   hipMemcpy(rstHstData, rstData, sizeof(float)*COUNT*BEI_SHU, hipMemcpyDeviceToHost);
	   */

	/*

	   for(int i=0; i<realCnt; i++)
	   {
	   printf("%.2f\n", rstHstData[i]);
	   }
	   */
}


