#include "hip/hip_runtime.h"
#ifdef _MSC_VER
typedef unsigned int uint;
#endif

#include <iostream>
#include <cstdio>
#include <cstdlib>

#include "globalDef.h"

//#include <windows.h>

#include "rasterPipe.h"
#include "__pikoCompiledPipe.h"

#ifdef __PIKOC_HOST__

#include <piko/builtinTypes.h>
#include "host_math.h"
#include "pikoTypes.h"
#include "rasterMacros.h"

#include "common_code/FPSMeter.h"

// pikoc does not work well with assimp, so it will not be included when pikoc runs
#ifndef __PIKOC__
#include "sceneParser.h" 
#endif  // __PIKOC__

#include <assert.h>


// useful macros
#define __DEB                     {printf("Reached %s (%d)\n",__FILE__,__LINE__);}
#define __ERR                     {printf("ERROR AT %s (%d)\n",__FILE__,__LINE__); exit(1);}
#define assertPrint(expr,msg,...) {if(!(expr)){printf("[ASSERT] "); printf(msg,__VA_ARGS__); assert(0);}}
#define raise(msg,...)            {assertPrint(0,msg,__VA_ARGS__);}

using namespace std;

#define TRIANGLE_BUFFER_SIZE (1024*1024*35)
//#define TRIANGLE_BUFFER_SIZE (1024*1024)

// ----------------------------------------
// function prototypes
// ----------------------------------------
void initScreen(int W, int H);
void initScene(int argc, char* argv[]);
void initPipe();
void display();
void destroyApp();
void doPerfTest(int n_runs = 10);
void mouseHandler(int button, int state, int x, int y);
void keypressed(unsigned char key, int x, int y);

// camera helper functions here
void buildProjectionMatrix();
void loadTriangleBuffer(int start, int end);
void resetDepthBuffer();
void findCameraZrange();

// ----------------------------------------
// global variables
// ----------------------------------------

#ifndef __PIKOC__
scene sMain;
#endif // __PIKOC__

// camera angles
float theta, phi, camdist;

raster_wtri* triangleBuffer = NULL;

// state
ConstantState pipelineConstantState;
MutableState pipelineMutableState;
int nTris, nVerts, nPatches;

//int Width = 1024, Height = 768;
int n_test_runs=0;

cvec3f bbmin = gencvec3f( FLT_MAX, FLT_MAX, FLT_MAX);
cvec3f bbmax = gencvec3f(-FLT_MAX,-FLT_MAX,-FLT_MAX);


// pipe
RasterPipe piko_pipe;

int main(int argc, char* argv[])
{
	//glutInit(&argc, argv);
	initScreen(SCREEN_WIDTH, SCREEN_HEIGHT);
	initScene(argc, argv);
	initPipe();
	display();
	//glutDisplayFunc(display);
	//doPerfTest(n_test_runs);
	//atexit(destroyApp);
	//glutMainLoop();
}

void initScreen(int W, int H)
{
#ifndef __PIKOC__
	sMain.cam().W() = W;
	sMain.cam().H() = H;
#endif // __PIKOC__

	pipelineConstantState.screenSizeX = W;
	pipelineConstantState.screenSizeY = H;

	pipelineConstantState.halfW = 0.5f * (float)W;
	pipelineConstantState.halfH = 0.5f * (float)H;

	//glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	//glutInitWindowSize(W,H);
	//glutCreateWindow("Raster Pipeline");
	//glutMouseFunc(mouseHandler);
	//glutKeyboardFunc(keypressed);
	//glClearColor(0.0f, 0.0f, 0.2f, 1.0f);
}


void initScene(int argc, char* argv[])
{
	// the scene will only be compiled when going through gcc and not pikoc
#ifndef __PIKOC__
	// parse scene file
	sceneParser scp;

	if(argc == 2)
	{
		n_test_runs = 0;
		scp.parseFile(".", argv[1], &sMain);
	}
	else if(argc == 3)
	{
		n_test_runs = atoi(argv[2]);
		scp.parseFile(".", argv[1], &sMain);
	}
	else
	{
		scp.parseFile(".", "fairyforest.scene", &sMain);
		n_test_runs = 100;
	}

	printf("Flattening scene assets: "); fflush(stdout);
	sMain.flatten(nTris, nVerts, nPatches);
	printf("T: %d, V: %d\n", nTris, nVerts, nPatches);

	//findCameraZrange();
	buildProjectionMatrix();

#endif // __PIKOC__
}

void initPipe()
{
	camera& cam = sMain.cam();

	// if(sMain.lights().size() > 0)
	// {
	//   pipelineState.lightPos = sMain.lights()[0].pos();
	//   pipelineState.lightColor = sMain.lights()[0].dif();
	// }
	// else
	// {
	//   // some default light that might suck
	//   pipelineState.lightPos = gencvec3f(1.0,1.0,1.0);
	//   pipelineState.lightColor = gencvec3f(1.0,1.0,1.0);
	// }

	loadTriangleBuffer(0, nTris);
	resetDepthBuffer();
	piko_pipe.allocate(pipelineConstantState, pipelineMutableState, triangleBuffer, nTris);
}

void display()
{
	printf("display()\n");
	// update state

	buildProjectionMatrix();
	resetDepthBuffer();

	piko_pipe.prepare();
	piko_pipe.run_single();

	unsigned* data =  piko_pipe.pikoScreen.getData();

	/*
	   glDrawPixels(pipelineConstantState.screenSizeX, pipelineConstantState.screenSizeY, GL_RGBA, GL_UNSIGNED_BYTE, data);
	   glutSwapBuffers();
	   */

	int x = pipelineConstantState.screenSizeX, y = pipelineConstantState.screenSizeY;
	FILE *fp = fopen("result", "w");
	for(int i=0; i<y; i++)
	{
		for(int j=0; j<x*4; j++)
		{
			fprintf(fp, "%d", ((char*)data)[i*x*4 + j]);
			if(j != x*4 - 1)
			{
				fprintf(fp, "\t");
			}
		}
		if(i != y - 1)
		{
			fprintf(fp, "\n");
		}
	}
	fclose(fp);

	return;

	// for(int i=0; i< pipelineState.screenSizeX * pipelineState.screenSizeY; i++)
	// {
	//   if(data[i] != 0)
	//     printf("%d: %x\n", i, data[i]);
	// }
}

void doPerfTest(int n_runs)
{
	printf("Running perf test (%d runs)...\n", n_runs);

	if(n_runs == 0) return;

	buildProjectionMatrix();
	resetDepthBuffer();
	piko_pipe.prepare();
	piko_pipe.run_single();

	Stopwatch mywatch;

	mywatch.Reset();
	for(int run = 0; run < n_runs; run++)
	{
		buildProjectionMatrix();
		resetDepthBuffer();
		piko_pipe.prepare();
	}
	float prepTime = mywatch.GetTime();

	mywatch.Reset();
	for(int run = 0; run < n_runs; run++)
	{
		buildProjectionMatrix();
		resetDepthBuffer();
		piko_pipe.prepare();
		piko_pipe.run_single();
	}
	float fullrunTime = mywatch.GetTime();

	float total_time_to_ms = 1000.0f / (float) n_runs;

	printf("Prep time     = %0.2f ms\n", total_time_to_ms * (prepTime));
	printf("Full run time = %0.2f ms\n", total_time_to_ms * (fullrunTime));
	printf("Raster time   = %0.2f ms\n", total_time_to_ms * (fullrunTime - prepTime));
}

void findCameraZrange()
{
	for(int i =0; i < sMain._flatnVertices; i++)
	{
		bbmin.x = min(bbmin.x, sMain._flattVertices[i].x);
		bbmin.y = min(bbmin.y, sMain._flattVertices[i].y);
		bbmin.z = min(bbmin.z, sMain._flattVertices[i].z);

		bbmax.x = max(bbmax.x, sMain._flattVertices[i].x);
		bbmax.y = max(bbmax.y, sMain._flattVertices[i].y);
		bbmax.z = max(bbmax.z, sMain._flattVertices[i].z);
	}

	camera& cam = sMain.cam();

	float d0 = magvec(cam.eye() - gencvec3f(bbmin.x, bbmin.y, bbmin.z));
	float d1 = magvec(cam.eye() - gencvec3f(bbmin.x, bbmin.y, bbmax.z));
	float d2 = magvec(cam.eye() - gencvec3f(bbmin.x, bbmax.y, bbmin.z));
	float d3 = magvec(cam.eye() - gencvec3f(bbmin.x, bbmax.y, bbmax.z));
	float d4 = magvec(cam.eye() - gencvec3f(bbmax.x, bbmin.y, bbmin.z));
	float d5 = magvec(cam.eye() - gencvec3f(bbmax.x, bbmin.y, bbmax.z));
	float d6 = magvec(cam.eye() - gencvec3f(bbmax.x, bbmax.y, bbmin.z));
	float d7 = magvec(cam.eye() - gencvec3f(bbmax.x, bbmax.y, bbmax.z));

	float mind = min(min(min(d0,d1),min(d2,d3)),min(min(d4,d5),min(d6,d7)));
	float maxd = max(max(max(d0,d1),max(d2,d3)),max(max(d4,d5),max(d6,d7)));

	cam.zNear() = max(0.01f, mind * 0.5f);
	cam.zFar() = min(maxd * 2.0f, mind * 10000.0f);

	printf("z range %f to %f\n", cam.zNear(), cam.zFar());

}


void getPerspectiveMat(float *mat, float fovy, 
		float aspect, float n, float f)
{
	float l, r, b, t;
	float pi = 4.0 * atan(1);
	float jiao = fovy * pi / 360;
	t = n * tanf(jiao);
	b = -t;
	l = b * aspect;
	r = t * aspect;

	memset(mat, 0, sizeof(float)*16);
	mat[0] = (2.0f * n) / (r-l);
	mat[2] = (r+l)/(r-l);
	mat[5] = (2.0f * n) / (t-b);
	mat[6] = (t+b)/(t-b);
	mat[10] = -(f+n)/(f-n);
	mat[11] = -(2.0f*f*n)/(f-n);
	mat[14] = -1.0;
}

void getLookUpMat(float *res, float ex, float ey, float ez,
		float tx, float ty, float tz, float ux, float uy, float uz)
{
	float fx = tx - ex;
	float fy = ty - ey;
	float fz = tz - ez;
	float flength = sqrt(fx*fx + fy*fy + fz*fz);
	fx /= flength;
	fy /= flength;
	fz /= flength;

	float sx, sy, sz;
	sx = fy*uz - fz*uy;
	sy = fz*ux - fx*uz;
	sz = fx*uy - fy*ux;
	float slength = sqrt(sx*sx + sy*sy + sz*sz);
	sx /= slength;
	sy /= slength;
	sz /= slength;

	ux = sy*fz - sz*fy;
	uy = sz*fx - sx*fz;
	uz = sx*fy - sy*fx;

	float mat[16];
	memset(mat, 0, sizeof(float)*16);
	mat[0] = 1;
	mat[5] = 1;
	mat[10] = 1;
	mat[15] = 1;

	mat[0] = sx;
	mat[1] = sy;
	mat[2] = sz;
	mat[4] = ux;
	mat[5] = uy;
	mat[6] = uz;
	mat[8] = -fx;
	mat[9] = -fy;
	mat[10] = -fz;

	float trans[16];
	memset(trans, 0, sizeof(float)*16);
	trans[0] = 1;
	trans[5] = 1;
	trans[10] = 1;
	trans[15] = 1;
	trans[3] = -ex;
	trans[7] = -ey;
	trans[11] = -ez;

	memset(res, 0, sizeof(float)*16);
	for(int i=0; i<4; i++)
	{
		for(int j=0; j<4; j++)
		{
			for(int k=0; k<4; k++)
			{
				res[i*4 + j] += mat[i*4 + k] * trans[k*4 + j];
			}
		}
	}


}



void buildProjectionMatrix()
{

	camera& cam = sMain.cam();
	float viewmat[16], mat[16], resProj[16], resView[16];

	//glMatrixMode(GL_PROJECTION);

	//glLoadIdentity();
	memset(viewmat, 0, sizeof(float)*16);
	viewmat[0] = 1;
	viewmat[5] = 1;
	viewmat[10] = 1;
	viewmat[15] = 1;
	getPerspectiveMat(mat, cam.fovyDeg(), cam.aspect(), cam.zNear(), cam.zFar());
	//gluPerspective(cam.fovyDeg(), cam.aspect(), cam.zNear(), cam.zFar());
	//glGetFloatv(GL_PROJECTION_MATRIX, viewmat);
	memset(resProj, 0, sizeof(resProj));
	for(int i=0; i<4; i++)
	{
		for(int j=0; j<4; j++)
		{
			for(int k=0; k<4; k++)
			{
				// res和viewmat是列优先，mat是行优先
				resProj[j*4+i] += viewmat[k*4+i]*mat[k*4+j];
			}
		}
	}
	//glLoadMatrixf(resProj);


	//glMatrixMode(GL_MODELVIEW);
	//glLoadIdentity();

	memset(viewmat, 0, sizeof(float)*16);
	viewmat[0] = 1;
	viewmat[5] = 1;
	viewmat[10] = 1;
	viewmat[15] = 1;

	float lu[16];
	getLookUpMat(lu, cam.eye().x,    cam.eye().y,    cam.eye().z,
			cam.target().x, cam.target().y, cam.target().z,
			cam.up().x,     cam.up().y,     cam.up().z);
	memset(resView, 0, sizeof(resView));
	for(int i=0; i<4; i++)
	{
		for(int j=0; j<4; j++)
		{
			for(int k=0; k<4; k++)
			{
				// res和viewmat是列优先，lu是行优先
				resView[j*4+i] += viewmat[k*4+i]*lu[k*4+j];
			}
		}
	}
	//glLoadMatrixf(resView);


	/*
	   gluLookAt(
	   cam.eye().x,    cam.eye().y,    cam.eye().z,
	   cam.target().x, cam.target().y, cam.target().z,
	   cam.up().x,     cam.up().y,     cam.up().z);
	   */

	/*
	   glGetFloatv(GL_MODELVIEW_MATRIX, viewmat);
	   for(int i=0; i<4; i++)
	   {
	   for(int j=0; j<4; j++)
	   {
	   printf("%.2f\t", viewmat[i*4+j]);
	   }
	   printf("\n");
	   }
	   printf("\n");
	   printf("\n");


	   for(int i=0; i<4; i++)
	   {
	   for(int j=0; j<4; j++)
	   {
	   printf("%.2f\t", lu[i*4+j]);
	   }
	   printf("\n");
	   }
	   printf("\n");
	   printf("\n");
	   */

			//glMatrixMode(GL_PROJECTION);
			//glPushMatrix();
			// glGetFloatv(GL_MODELVIEW_MATRIX, pipelineConstantState.viewMatrix);
			memcpy(pipelineConstantState.viewMatrix, resView, sizeof(float)*16);
		//glMultMatrixf(pipelineConstantState.viewMatrix);
		float newProj[16];
		memset(newProj, 0, sizeof(newProj));
		for(int i=0; i<4; i++)
		{
			for(int j=0; j<4; j++)
			{
				for(int k=0; k<4; k++)
				{
					newProj[j*4 + i] += resProj[k*4+i] * resView[j*4 + k];
				}
			}
		}
		memcpy(pipelineConstantState.viewProjMatrix, newProj, sizeof(float)*16);

		//glGetFloatv(GL_PROJECTION_MATRIX, pipelineConstantState.viewProjMatrix);
		//glPopMatrix();  


		// printf("final projection matrix:\n");
		// for(int i=0; i<16; i++) {
		//   if (i%4 ==0) printf("\n");
		//     printf("%f ", pipelineConstantState.viewProjMatrix[i]);
		// } printf("\n");

		//   printf("final modelview matrix:\n");
		// for(int i=0; i<16; i++) {
		//   if (i%4 ==0) printf("\n");
		//     printf("%f ", pipelineConstantState.viewMatrix[i]);
		// } printf("\n");
}

/*
   void buildProjectionMatrix()
   {
   camera& cam = sMain.cam();

   glMatrixMode(GL_PROJECTION); 

	//printf("znear = %f, zfar = %f\n", cam.zNear(), cam.zFar());

	glLoadIdentity();
	gluPerspective(cam.fovyDeg(), cam.aspect(), cam.zNear(), cam.zFar());
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	gluLookAt(
	cam.eye().x,    cam.eye().y,    cam.eye().z,
	cam.target().x, cam.target().y, cam.target().z,
	cam.up().x,     cam.up().y,     cam.up().z);

	glMatrixMode(GL_PROJECTION);
	glPushMatrix();
	glGetFloatv(GL_MODELVIEW_MATRIX, pipelineConstantState.viewMatrix);
	glMultMatrixf( pipelineConstantState.viewMatrix);
	glGetFloatv(GL_PROJECTION_MATRIX, pipelineConstantState.viewProjMatrix);
	glPopMatrix();

	// printf("final projection matrix:\n");
	// for(int i=0; i<16; i++) {
	//   if (i%4 ==0) printf("\n");
	//     printf("%f ", pipelineState.viewProjMatrix[i]);
	// } printf("\n");
	}
	*/

void inPlaceTransform(cvec3f& v, float viewmat[16])
{
	cvec4f tv;
	vtransform(viewmat, v, tv);
	float onebyw = 1.0f / tv.w;
	tv.x = (tv.x * onebyw + 1.0f) * pipelineConstantState.halfW;
	tv.y = (tv.y * onebyw + 1.0f) * pipelineConstantState.halfH;
	tv.z = (tv.z * onebyw );
	v = gencvec3f(tv.x, tv.y, tv.z);
}

inline void saturatePixelHost(cvec3f& _p) 
{
	_p.x = _p.x > 1.0f ? 1.0f : (_p.x < 0.0f ? 0.0f : _p.x);
	_p.y = _p.y > 1.0f ? 1.0f : (_p.y < 0.0f ? 0.0f : _p.y);
	_p.z = _p.z > 1.0f ? 1.0f : (_p.z < 0.0f ? 0.0f : _p.z);
}

inline cvec3f computeLightingHost(const cvec3f& _mynor, cvec3f& _lightvec, cvec3f& _matcol)
{
	cvec3f out;
	float _diffuse = 
		_mynor.x * _lightvec.x + _mynor.y * _lightvec.y + _mynor.z * _lightvec.z; 
	_diffuse = _diffuse < 0.0f ? 0.0f : (_diffuse );
	out.x = (_diffuse * _matcol.x + 0.15f); 
	out.y = (_diffuse * _matcol.y + 0.15f); 
	out.z = (_diffuse * _matcol.z + 0.30f); 
	saturatePixelHost(out);
	return out;
}

inline unsigned toABGRHost(cvec3f color)
{
	//printf("r = %f\n", color.x);
	return ((255<<24) | ((unsigned)(color.z*255.0f)<<16) | ((unsigned)(color.y*255.0f)<<8) | (unsigned)(color.x*255.0f));
}

void loadTriangleBuffer(int start, int end)
{

#ifdef VTX_PRETRANSFORM
	printf("Pretransforming vertices\n");
#endif

	printf("111111\n");
	fflush(NULL);

	// lazy create
	if(triangleBuffer == NULL)
	{
		triangleBuffer = new raster_wtri[TRIANGLE_BUFFER_SIZE];
	}


	printf("222222\n");
	fflush(NULL);

	int size = end - start;

	if (size <=0) return;

	int counter = 0;
	for(int i=start; i<end; i++)
	{
		int t0 = sMain._flatTriangles[i].x;
		int t1 = sMain._flatTriangles[i].y;
		int t2 = sMain._flatTriangles[i].z;
		triangleBuffer[counter].worldPos0 = gencvec3f(sMain._flattVertices[t0].x, sMain._flattVertices[t0].y, sMain._flattVertices[t0].z);
		triangleBuffer[counter].worldPos1 = gencvec3f(sMain._flattVertices[t1].x, sMain._flattVertices[t1].y, sMain._flattVertices[t1].z);
		triangleBuffer[counter].worldPos2 = gencvec3f(sMain._flattVertices[t2].x, sMain._flattVertices[t2].y, sMain._flattVertices[t2].z);

		sMain._flattNormals[t0] = HOST::normalize(sMain._flattNormals[t0]);
		sMain._flattNormals[t1] = HOST::normalize(sMain._flattNormals[t1]);
		sMain._flattNormals[t2] = HOST::normalize(sMain._flattNormals[t2]);

#ifdef VTX_PRETRANSFORM
		cvec3f lightvec0 = (sMain.cam().eye() - triangleBuffer[counter].worldPos0);
		cvec3f lightvec1 = (sMain.cam().eye() - triangleBuffer[counter].worldPos1);
		cvec3f lightvec2 = (sMain.cam().eye() - triangleBuffer[counter].worldPos2);

		normalizeInplace(lightvec0);
		normalizeInplace(lightvec1);
		normalizeInplace(lightvec2);

		inPlaceTransform(triangleBuffer[counter].worldPos0, pipelineConstantState.viewProjMatrix);
		inPlaceTransform(triangleBuffer[counter].worldPos1, pipelineConstantState.viewProjMatrix);
		inPlaceTransform(triangleBuffer[counter].worldPos2, pipelineConstantState.viewProjMatrix);

		cvec3f matcol   = gencvec3f(0.9000f, 0.9000f, 0.6000f);
		cvec3f lightvec = gencvec3f(0.5773f, 0.5773f, 0.5773f);


		triangleBuffer[counter].icol0 = toABGRHost(computeLightingHost(sMain._flattNormals[t0], lightvec0, matcol));
		triangleBuffer[counter].icol1 = toABGRHost(computeLightingHost(sMain._flattNormals[t1], lightvec1, matcol));
		triangleBuffer[counter].icol2 = toABGRHost(computeLightingHost(sMain._flattNormals[t2], lightvec2, matcol));
#else
		triangleBuffer[counter].normal0 = gencvec2f(sMain._flattNormals[t0].x, sMain._flattNormals[t0].y);
		triangleBuffer[counter].normal1 = gencvec2f(sMain._flattNormals[t1].x, sMain._flattNormals[t1].y);
		triangleBuffer[counter].normal2 = gencvec2f(sMain._flattNormals[t2].x, sMain._flattNormals[t2].y);
#endif
		triangleBuffer[counter].id = counter;
		counter++;
	}

	printf("Added %d triangles\n", counter);

	// printf("bounds are %f %f %f to %f %f %f\n",
	//   bbmin.x, bbmin.y, bbmin.z,
	//   bbmax.x, bbmax.y, bbmax.z);
}

void resetDepthBuffer()
{
	int nPixels = pipelineConstantState.screenSizeX * pipelineConstantState.screenSizeY;
	for(int i = 0; i < nPixels; i++)
	{
		pipelineMutableState.zBuffer[i] = 1.0f;
	}
}

void destroyApp()
{
	piko_pipe.destroy();
}

/*
   void mouseHandler(int button, int state, int x, int y) {
   if(button == GLUT_LEFT_BUTTON && state == GLUT_DOWN) {
	//onMouse = 1;
	printf("Debugging pixel %d %d\n", x, y);
	pipelineConstantState.debX = x;
	pipelineConstantState.debY = Height - y;

	glutPostRedisplay();
	}
	}

	void keypressed(unsigned char key, int x, int y)
	{
	switch(key)
	{
	case 27: exit(0); break;
	}
	}

*/

#endif // __PIKOC_HOST__
