#include "hip/hip_runtime.h"
#include "bezmesh.h"

// tokenize a string based on a set of single-char delimiters
inline void bezmesh::vtokenize(const std::string& str,const std::string& delimiters, std::vector<std::string> &tokens)
{
    tokens.clear();
    // if empty, return empty
    if(str=="") return;
    // skip delimiters at beginning.
    std::string::size_type lastPos = str.find_first_not_of(delimiters, 0);
    // find first "non-delimiter".
    std::string::size_type pos = str.find_first_of(delimiters, lastPos);
    while (std::string::npos != pos || std::string::npos != lastPos)
    {
        // found a token, add it to the vector.
        tokens.push_back(str.substr(lastPos, pos - lastPos));
        // skip delimiters.  Note the "not_of"
        lastPos = str.find_first_not_of(delimiters, pos);
        // find next "non-delimiter"
        pos = str.find_first_of(delimiters, lastPos);
    }
}
int bezmesh::read_bez(const char* filename, float scale=1.0f, bool swapyz){//{{{

  FILE *f;

  char line[500];
  string sline;
  vector<string> tokens;
  string delim=" \t\n\r";

  f=fopen(filename,"r");

  if(!f)
  {
    printf("File %s not found\n",filename);
    return -1;
  }

  printf("Reading mesh...\n");fflush(stdout);

  while(fgets(line, 500, f))
  {
    sline = line;
    vtokenize(sline, delim, tokens);
    //printf("%s \n=\n",sline.c_str());
    if(tokens.size()>=2){
      if(tokens[0]=="Patch" && tokens[1]=="\"bicubic\""){
        // The next 4 lines contain 16 points.
        patch p;
        for(int i=0; i<4; i++){
          if(fgets(line, 500, f)){
            sline = line;
            vtokenize(sline, delim, tokens);
            for(int j=0; j<4; j++){
							p.cp[i][j].x = (float)strtod(tokens[j*3+0].c_str(),NULL);
							p.cp[i][j].y = (float)strtod(tokens[j*3+1].c_str(),NULL);
							p.cp[i][j].z = (float)strtod(tokens[j*3+2].c_str(),NULL);
              if(swapyz) swap(p.cp[i][j].y, p.cp[i][j].z);
							p.cp[i][j].w = 1.0f;

              //printf("Point %d %d: %0.2f %0.2f %0.2f %0.2f\n",i,j,p.cp[i][j].x, p.cp[i][j].y, p.cp[i][j].z, p.cp[i][j].w);
            }
          }else{
            return -1;
          }
        }
				p.parent = this;
        _patches.push_back(p);
      }
    }
  }


  if(scale!=0.0f) patches_normalize(scale);

  fclose(f);
  return (int)(_patches.size()*16);
  //return 32;
}//}}}

float* bezmesh::get_patches(){//{{{

  float *control_points;
  // each patch is 16 control points, i.e. 16x4 floats
  control_points = new float[_patches.size() * 16 * 4];

  for(int i=0; i<(int)_patches.size(); i++){
    for(int j=0; j<4; j++){
      for(int k=0; k<4; k++){
        *(control_points + (i*16 + j*4 + k)*4 + 0) = _patches[i].cp[j][k].x;
        *(control_points + (i*16 + j*4 + k)*4 + 1) = _patches[i].cp[j][k].y;
        *(control_points + (i*16 + j*4 + k)*4 + 2) = _patches[i].cp[j][k].z;
        *(control_points + (i*16 + j*4 + k)*4 + 3) = _patches[i].cp[j][k].w;
      }
    }
  }

  //for(int i=0; i<(int)_patches.size()*16*4; i++){
  //	printf("%0.2f\t",*(control_points + i));
  //}

  return control_points;
}//}}}

void bezmesh::get_patches(float * control_points){ //{{{

	// each patch is 16 control points, i.e. 16x4 floats
	for(int i=0; i<(int)_patches.size(); i++){
		for(int j=0; j<4; j++){
			for(int k=0; k<4; k++){
				*(control_points + (i*16 + j*4 + k)*4 + 0) = _patches[i].cp[j][k].x;
				*(control_points + (i*16 + j*4 + k)*4 + 1) = _patches[i].cp[j][k].y;
				*(control_points + (i*16 + j*4 + k)*4 + 2) = _patches[i].cp[j][k].z;
				*(control_points + (i*16 + j*4 + k)*4 + 3) = _patches[i].cp[j][k].w;
			}
		}
	}

	//for(int i=0; i<(int)_patches.size()*16*4; i++){
	//	printf("%0.2f\t",*(control_points + i));
	//}

}//}}}

patch& bezmesh::get_patches(int i){/////////////////////added by Xin 2010.11.17 {{{
	return _patches[i];
}//}}}

void bezmesh::patches_normalize(float scale){ //{{{

  float maxcoord=0.0f;

  printf("%ld patches\n",_patches.size());

  for(int i=0; i<(int)_patches.size(); i++){
    for(int j=0; j<4; j++){
      for(int k=0; k<4; k++){

        if(fabsf(_patches[i].cp[j][k].x) > maxcoord) maxcoord = _patches[i].cp[j][k].x;
        if(fabsf(_patches[i].cp[j][k].y) > maxcoord) maxcoord = _patches[i].cp[j][k].y;
        if(fabsf(_patches[i].cp[j][k].z) > maxcoord) maxcoord = _patches[i].cp[j][k].z;
      }
    }
  }

  float factor = scale/maxcoord;

  printf("factor = %f\n",factor);

  for(int i=0; i<(int)_patches.size(); i++){
    for(int j=0; j<4; j++){
      for(int k=0; k<4; k++){

        _patches[i].cp[j][k].x*=factor;
        _patches[i].cp[j][k].y*=factor;
        _patches[i].cp[j][k].z*=factor;
      }
    }
  }
}//}}}

cvec4f patch::pos_at_uv(float u, float v){ //{{{
  cvec4f point;
  for(int i=0; i<4; i++){
    for(int j=0; j<4; j++){
      point += (cp[i][j]*B(i,u)*B(j,v));
    }
  }
  return point;
}//}}}

cvec3f patch::nor_at_uv(float u, float v){ //{{{
  cvec3f tangu, tangv;
  for(int i=0; i<4; i++){
    for(int j=0; j<4; j++){
      tangu += (gencvec3f(cp[i][j].x,cp[i][j].y,cp[i][j].z)*dB(i,u)*B(j,v));
    }
  }
  for(int i=0; i<4; i++){
    for(int j=0; j<4; j++){
      tangv += (gencvec3f(cp[i][j].x,cp[i][j].y,cp[i][j].z)*B(i,u)*dB(j,v));
    }
  }
  cvec3f norm=cross(tangu, tangv);
  normalizeInplace(norm);
  return norm;
} //}}}
