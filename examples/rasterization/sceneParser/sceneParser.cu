#include "hip/hip_runtime.h"
#ifdef WIN32
typedef unsigned int uint;
#endif

//#include <windows.h>


#include "sceneParser.h"

static float viewmat[16];

template<typename to, typename from>
inline to lexical_cast(from const &x) {
    std::stringstream os;
    to ret;
    os << x;
    os >> ret;
    return ret;
}

inline void chompString(std::string& str){
    std::string::size_type pos = str.find_last_not_of("\n\r");
    if(pos != std::string::npos) {
        str.erase(pos + 1);
        pos = str.find_first_not_of("\n\r");
        if(pos != std::string::npos) str.erase(0, pos);
    }
    else str.erase(str.begin(), str.end());
}

inline void trimString(std::string& str){
    std::string::size_type pos = str.find_last_not_of(' ');
    if(pos != std::string::npos) {
        str.erase(pos + 1);
        pos = str.find_first_not_of(' ');
        if(pos != std::string::npos) str.erase(0, pos);
    }
    else str.erase(str.begin(), str.end());
}

// tokenize a string based on a set of single-char delimiters
inline void ltokenize(const std::string& str,const std::string& delimiters, std::list<std::string> &tokens)
{
    tokens.clear();
    // if empty, return empty
    if(str=="") return;
    // skip delimiters at beginning.
    std::string::size_type lastPos = str.find_first_not_of(delimiters, 0);
    // find first "non-delimiter".
    std::string::size_type pos = str.find_first_of(delimiters, lastPos);
    while (std::string::npos != pos || std::string::npos != lastPos)
    {
        // found a token, add it to the vector.
        tokens.push_back(str.substr(lastPos, pos - lastPos));
        // skip delimiters.  Note the "not_of"
        lastPos = str.find_first_not_of(delimiters, pos);
        // find next "non-delimiter"
        pos = str.find_first_of(delimiters, lastPos);
    }
}


inline void vtokenize_degen(const std::string& str,const std::string& delimiters, std::vector<std::string> &tokens)
{
    using namespace std;
    tokens.clear();
    string::size_type delimPos = 0, tokenPos = 0, pos = 0;

    if(str.length()<1)  return;
    while(1){
        delimPos = str.find_first_of(delimiters, pos);
        tokenPos = str.find_first_not_of(delimiters, pos);

        if(string::npos != delimPos){
            if(string::npos != tokenPos){
                if(tokenPos<delimPos){
                    tokens.push_back(str.substr(pos,delimPos-pos));
                }else{
                    tokens.push_back("");
                }
            }else{
                tokens.push_back("");
            }
            pos = delimPos+1;
        } else {
            if(string::npos != tokenPos){
                tokens.push_back(str.substr(pos));
            } else {
                tokens.push_back("");
            }
            break;
        }
    }
}


// tokenize a string based on a set of single-char delimiters
inline void vtokenize(const std::string& str,const std::string& delimiters, std::vector<std::string> &tokens)
{
    tokens.clear();
    // if empty, return empty
    if(str=="") return;
    // skip delimiters at beginning.
    std::string::size_type lastPos = str.find_first_not_of(delimiters, 0);
    // find first "non-delimiter".
    std::string::size_type pos = str.find_first_of(delimiters, lastPos);
    while (std::string::npos != pos || std::string::npos != lastPos)
    {
        // found a token, add it to the vector.
        tokens.push_back(str.substr(lastPos, pos - lastPos));
        // skip delimiters.  Note the "not_of"
        lastPos = str.find_first_not_of(delimiters, pos);
        // find next "non-delimiter"
        pos = str.find_first_of(delimiters, lastPos);
    }
}

bool sceneParser::fetchLine(){
    if(sceneFile.good()){
        getline(sceneFile, curLine);
        return true;
    }else
        return false;
}

void sceneParser::processLine()
{
    // process a line freshly

    // trim any extra spaces
    trimString(curLine);

    // eliminate comments
    string::size_type pos = curLine.find_first_of('#');
    string noCommentsLine = curLine.substr(0, pos);

    // tokenize
    ltokenize(noCommentsLine, " \t", curTokens);

    //test
    //if(curTokens.size()>0) printf("Line: {%s}\n", noCommentsLine.c_str());
}

bool sceneParser::fetchNextToken(string& token){
    // if you don't have any tokens,
    // fetch next line
    while(1){
        if(curTokens.size()>0){
            break;
        }else{
            if(fetchLine()) processLine();
            else            return false;
        }
    }

    // the first token should be the command
    token = curTokens.front();
    curTokens.pop_front();

    //printf("fetched %s\n", token.c_str());
    return true;
}

bool sceneParser::fetchLeftBrace(){
    string b;
    if(fetchNextToken(b) && b=="["){
        return true;
    }else{
        printf("\tCannot fetch '['\n");
        return false;
    }
}

bool sceneParser::fetchRightBrace(){
    string b;
    if(fetchNextToken(b) && b=="]"){
        return true;
    }else{
        printf("\tCannot fetch ']'\n");
        return false;
    }
}

bool sceneParser::fetchString(string& s){
    if(fetchNextToken(s)){
        return true;
    }else{
        return false;
    }
}

bool sceneParser::fetch1f(float& x){
    string sx;
    if(fetchNextToken(sx)){
        x = lexical_cast<float, string>(sx);
        return true;
    }else{
        printf("\tCannot fetch 1f\n");
        return false;
    }
}

bool sceneParser::fetch2f(float& x, float& y){
    if( fetchLeftBrace() && fetch1f(x) && fetch1f(y) && fetchRightBrace()){
        return true;
    }else{
        printf("\tCannot fetch 2f\n");
        return false;
    }
}

bool sceneParser::fetch3f(float& x, float& y, float& z){
    if( fetchLeftBrace() && fetch1f(x) && fetch1f(y) && fetch1f(z) && fetchRightBrace()){
        return true;
    }else{
        printf("\tCannot fetch 3f\n");
        return false;
    }
}

bool sceneParser::fetchCamera(){
    float ex, ey, ez;
    float ox, oy, oz;
    float ux, uy, uz;

    printf("[camera]\n"); //fflush(stdout);
    if( fetch3f(ex, ey, ez) && fetch3f(ox, oy, oz) && fetch3f(ux, uy, uz) ){
		curScene->cam().updateCam(gencvec3f(ex, ey, ez), gencvec3f(ox, oy, oz), gencvec3f(ux, uy, uz), PI/3.0f, 1.0f);
        // fetch aperture etc:
        float aperture, foc_length, foc_plane;
        if( fetch3f(aperture, foc_length, foc_plane) ){
            curScene->cam().aperture()    = aperture;
            curScene->cam().focallength() = foc_length;
            curScene->cam().focalplane()  = foc_plane;
        }else{
            printf("\t[pinhole]\n"); //fflush(stdout);
            curScene->cam().aperture()    = 0.0f;
            curScene->cam().focallength() = 1.0f;
            curScene->cam().focalplane()  = 1.0f;
        }
        return true;
    }else return false;
}

bool sceneParser::fetchLight(){
    float px, py, pz;
    float dr, dg, db;
    float sr, sg, sb, sn;

    printf("[light]\n"); //fflush(stdout);
    if( fetch3f(px, py, pz) && fetch3f(dr, dg, db) && fetch3f(sr, sg, sb) && fetch1f(sn))
    {
        curScene->lights().push_back(light(gencvec3f(px,py,pz),gencvec3f(dr,dg,db),gencvec3f(0.0f,0.0f,0.0f),gencvec3f(sr,sg,sb),sn));
        return true;
    }else
        return false;
}

bool sceneParser::fetchMaterial(){
    float ar, ag, ab;
    float dr, dg, db;
    float sr, sg, sb, sn;

    printf("[material]\n"); //fflush(stdout);
    if( fetch3f(ar, ag, ab) && fetch3f(dr, dg, db) && fetch3f(sr, sg, sb) && fetch1f(sn)){
        curMat = material(MAT_DIFFUSE, gencvec3f(dr,dg,db), gencvec3f(ar,ag,ab), gencvec3f(sr,sg,sb), sn);
        curScene->mats().push_back(curMat);
        return true;
    }else return false;
}

bool sceneParser::fetchMeshAttribute(){
    string attrname;
    float  attrval;

    if( fetchString(attrname) && fetch1f(attrval)){
        //printf("[attribute] %s\n",attrname.c_str());
        curAttrs[attrname] = (int)attrval;
        return true;
    } else {
        //printf("[attribute]\n"); fflush(stdout);
        return false;
    }

}

void getTranslateMat(float *tm, float rx, float ry, float rz)
{
	memset(tm, 0, sizeof(float)*16);
	tm[0] = 1;
	tm[3] = rx;
	tm[5] = 1;
	tm[7] = ry;
	tm[10] = 1;
	tm[11] = rz;
	tm[15] = 1;
}

bool sceneParser::fetchTranslate(){
    //printf("[translate]\n"); fflush(stdout);
    float tx, ty, tz;
    if( fetch3f(tx, ty, tz) ){
		float tmat[16], res[16];
        //glGetFloatv(GL_MODELVIEW_MATRIX, viewmat);
		
		/*
		for(int i=0; i<4; i++)
		{
			for(int j=0; j<4; j++)
			{
				printf("%.2f\t", viewmat[i*4+j]);
			}
			printf("\n");
		}
		printf("\n");
		printf("\n");
		
        glTranslatef(tx, ty, tz);
		
        glGetFloatv(GL_MODELVIEW_MATRIX, viewmat);
		for(int i=0; i<4; i++)
		{
			for(int j=0; j<4; j++)
			{
				printf("%.2f\t", viewmat[i*4+j]);
			}
			printf("\n");
		}
		printf("\n");
		printf("\n");
		
		printf("%f, %f, %f\n", tx, ty, tz);
			
		getTranslateMat(tmat, tx, ty, tz);
		for(int i=0; i<4; i++)
		{
			for(int j=0; j<4; j++)
			{
				printf("%.2f\t", tmat[i*4+j]);
			}
			printf("\n");
		}
		*/
		
		getTranslateMat(tmat, tx, ty, tz);
		memset(res, 0, sizeof(res));
		for(int i=0; i<4; i++)
		{
			for(int j=0; j<4; j++)
			{
				for(int k=0; k<4; k++)
				{
					// res和viewmat是列优先，tmat是行优先
					res[j*4+i] += viewmat[k*4+i]*tmat[k*4+j];
				}
			}
		}
		//glLoadMatrixf(res);
		memcpy(viewmat, res, sizeof(float)*16);
		
        return true;
    }else return false;
}

void getRotateMat(float *rotatemat, float rt, 
		float rx, float ry, float rz)
{
	memset(rotatemat, 0, sizeof(float)*16);
	rotatemat[15] = 1;
	if((int)(rx + ry + rz) != 1)
	{
		printf("%s in %s, at line %d\n", __FUNCTION__, __FILE__, __LINE__);
		return;
	}
	else if(abs(rx - 1.0f) < 1e-5)
	{
		printf("%s in %s, at line %d\n", __FUNCTION__, __FILE__, __LINE__);
		rotatemat[0] = 1;
		rotatemat[5] = cos(rt);
		rotatemat[6] = -sin(rt);
		rotatemat[9] = sin(rt);
		rotatemat[10] = cos(rt);
	}
	else if(abs(ry - 1.0f) < 1e-5)
	{
		printf("%s in %s, at line %d\n", __FUNCTION__, __FILE__, __LINE__);
		rotatemat[0] = cos(rt);
		rotatemat[2] = sin(rt);
		rotatemat[5] = 1;
		rotatemat[8] = -sin(rt);
		rotatemat[10] = cos(rt);
	}
	else if(abs(rz - 1.0f) < 1e-5)
	{
		printf("%s in %s, at line %d\n", __FUNCTION__, __FILE__, __LINE__);
		rotatemat[0] = cos(rt);
		rotatemat[1] = -sin(rt);
		rotatemat[4] = sin(rt);
		rotatemat[5] = cos(rt);
		rotatemat[10] = 1;
	}
	else
	{
		printf("%s in %s, at line %d\n", __FUNCTION__, __FILE__, __LINE__);
		return;
	}
}

bool sceneParser::fetchRotate(){
    //printf("[rotate]\n"); fflush(stdout);
    float rx, ry, rz, rt;
    if( fetch3f(rx, ry, rz) && fetch1f(rt) ){
		// glRotatef(rt, rx, ry, rz);
		
		float rotatemat[16], res[16];
        //glGetFloatv(GL_MODELVIEW_MATRIX, viewmat);
		float pi = 4.0 * atan(1);
		getRotateMat(rotatemat, rt/180*pi, rx, ry, rz);
		
		memset(res, 0, sizeof(res));
		for(int i=0; i<4; i++)
		{
			for(int j=0; j<4; j++)
			{
				for(int k=0; k<4; k++)
				{
					// res和viewmat是列优先，rotatemat是行优先
					res[j*4+i] += viewmat[k*4+i]*rotatemat[k*4+j];
				}
			}
		}
		//glLoadMatrixf(res);

		memcpy(viewmat, res, sizeof(float)*16);
		
        return true;
    }else return false;
}


void getScaleMat(float *sm, float rx, float ry, float rz)
{
	memset(sm, 0, sizeof(float) * 16);
	sm[0] = rx;
	sm[5] = ry;
	sm[10] = rz;
	sm[15] = 1;
}

bool sceneParser::fetchScale(){
    //printf("[scale]\n"); fflush(stdout);
    float sx, sy, sz;
    if( fetch3f(sx, sy, sz) ){
        //glScalef(sx,sy,sz);
		float smat[16], res[16];
        //glGetFloatv(GL_MODELVIEW_MATRIX, viewmat);
		getScaleMat(smat, sx, sy, sz);
		memset(res, 0, sizeof(res));
		for(int i=0; i<4; i++)
		{
			for(int j=0; j<4; j++)
			{
				for(int k=0; k<4; k++)
				{
					// res和viewmat是列优先，smat是行优先
					res[j*4+i] += viewmat[k*4+i]*smat[k*4+j];
				}
			}
		}
		//glLoadMatrixf(res);
		memcpy(viewmat, res, sizeof(float)*16);
        return true;
    }else return false;
}

bool sceneParser::fetchTexture(){
     //printf("[texture]\n"); fflush(stdout);
     string texfile;

     if( fetchString(texfile)){
         //curScene->addTexture("textures/" + texfile);
         return true;

     }else return false;
}

bool sceneParser::fetchLoadbez()
{
	printf("[loadbez]\n"); fflush(stdout);

	string meshfile;

    if( fetchString(meshfile)){
        meshfile = basepath_ + "/meshes/" + meshfile;
        bezmesh *m = new bezmesh(meshfile);
		if(curScene->mats().size()==0)
             curScene->mats().push_back(curMat);

		 m->matID()      = curScene->mats().size()-1;
         m->attributes() = curAttrs;

         //apply current transformation to m

         float invviewmat[16];
         //glGetFloatv(GL_MODELVIEW_MATRIX, viewmat);
         GenerateInverseMatrix4f(invviewmat, viewmat);

         m->applyTransformation(viewmat, invviewmat);

         curScene->addBezmesh(m);

		return true;
	}
	return false;
}

bool sceneParser::fetchLoadmesh(){
     printf("\t[loadmesh]\n"); fflush(stdout);
     string meshfile;

     if( fetchString(meshfile)){
         meshfile = basepath_ + "/meshes/" + meshfile;
         trimesh *m = new trimesh(meshfile);
         //printf("loading %s\n", meshfile.c_str());
         if(curScene->mats().size()==0)
             curScene->mats().push_back(curMat);

         m->matID()      = curScene->mats().size()-1;
         m->attributes() = curAttrs;

         //apply current transformation to m

         float invviewmat[16];
         //glGetFloatv(GL_MODELVIEW_MATRIX, viewmat);
         GenerateInverseMatrix4f(invviewmat, viewmat);

         m->applyTransformation(viewmat, invviewmat);

         curScene->addMesh(m);

         return true;
     }else return false;
}

bool sceneParser::fetchReset(){
     //printf("[reset]\n"); fflush(stdout);
     //glMatrixMode(GL_MODELVIEW);
     //glLoadIdentity();
	 memset(viewmat, 0, sizeof(float)*16);
	viewmat[0] = 1;
	viewmat[5] = 1;
	viewmat[10] = 1;
	viewmat[15] = 1;

     return true;
}

bool sceneParser::fetchCommand(){

    string cmd;

    if(!fetchNextToken(cmd)){
        return false;
    }

    if      (cmd == "camera"){          return fetchCamera();
    }else if(cmd == "camera_fov"){      return fetchCameraFov();
    }else if(cmd == "zNear"){           return fetchZnear();
    }else if(cmd == "zFar"){            return fetchZfar();
    }else if(cmd == "light"){           return fetchLight();
    }else if(cmd == "material"){        return fetchMaterial();
    }else if(cmd == "translate"){       return fetchTranslate();
    }else if(cmd == "rotate"){          return fetchRotate();
    }else if(cmd == "scale"){           return fetchScale();
    }else if(cmd == "loadmesh"){        return fetchLoadmesh();
	  }else if(cmd == "loadbez"){         return fetchLoadbez();
    }else if(cmd == "reset"){           return fetchReset();
    }else if(cmd == "meshAttribute"){   return fetchMeshAttribute();
    }else if(cmd == "texture"){         return fetchTexture();
    }else{
      printf("Found dangling %s\n",cmd.c_str());
      return false;
    }

    curTokens.clear();
    return true;

}

bool sceneParser::fetchCameraFov(){
  float fov;
  printf("[camera_fov]\n"); //fflush(stdout);
  if(fetch1f(fov)){
    curScene->cam()._fovy = fov * PI/180.0f;
    return true;
  }else{
    printf("\tCannot fetch fov\n");
    return false;
  }

}

bool sceneParser::fetchZnear(){
  float znear; printf("[znear]\n");
  if(fetch1f(znear)){ curScene->cam().zNear() = znear;
    return true;
  }else{
    printf("\tCannot fetch znear\n");
    return false;
  }

}

bool sceneParser::fetchZfar(){
  float zfar; printf("[zfar]\n");
  if(fetch1f(zfar)){ curScene->cam().zFar() = zfar;
    return true;
  }else{
    printf("\tCannot fetch zfar\n");
    return false;
  }

}

void sceneParser::parseFile(string basepath, string filename, scene* sc)
{

    basepath_ = basepath;
    filename = basepath_ + "/scenes/" + filename;

    printf("Reading %s\n",filename.c_str());

    sceneFile.open(filename.c_str());

    if(!sceneFile.is_open()){
        printf("Cannot open %s\n", filename.c_str());
        return;
    }

    curScene = sc;


    //glMatrixMode(GL_MODELVIEW);
    //glPushMatrix();
    //glLoadIdentity();
	
	memset(viewmat, 0, sizeof(float)*16);
	viewmat[0] = 1;
	viewmat[5] = 1;
	viewmat[10] = 1;
	viewmat[15] = 1;

    while(fetchCommand());

    //glPopMatrix();
    printf("\n");
}

